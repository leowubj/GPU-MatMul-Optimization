#include "hip/hip_runtime.h"
// ;-*- mode: c;-*-
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "../src/utils.h"
#include "../src/types.h"
#include "mytypes.h"
using namespace std;

#include <stdio.h>

#define a(i, j, ld) a[ (i)*(ld) + (j) ]
#define b(i, j, ld) b[ (i)*(ld) + (j) ]
#define c(i, j, ld) c[ (i)*(ld) + (j) ]

#ifdef NAIVE
__global__ void matMul(int N, _FTYPE_ *C, _FTYPE_ *A, _FTYPE_ *B) {

    int I =  blockIdx.y*blockDim.y + threadIdx.y;
    int J =  blockIdx.x*blockDim.x + threadIdx.x;

    if((I < N) && (J < N)){
        _FTYPE_ _c = 0;
        for (unsigned int k = 0; k < N; k++) {
            _FTYPE_ a = A[I * N + k];
            _FTYPE_ b = B[k * N + J];
            _c += a * b;
        }
        C[I * N + J] = _c;
    }
}

#else
//You should be changing the kernel here for the non naive implementation.

//Shared memory
extern __shared__ _FTYPE_ smem[];

// matMul with only shared memory
/* __global__ void matMul(int N, _FTYPE_ *C, _FTYPE_ *A, _FTYPE_ *B) {
  int ty = threadIdx.y;
  int tx = threadIdx.x;
  int by = blockIdx.y;
  int bx = blockIdx.x;

  int abs_row_idx =  by*blockDim.y + ty;
  int abs_col_idx =  bx*blockDim.x + tx;

  _FTYPE_ * __restrict__ As = &smem[0];
  _FTYPE_ * __restrict__ Bs = &smem[TILEDIM_M * TILEDIM_M];
  _FTYPE_ Cij = 0;

  
  #pragma unroll
  for (int kk = 0; kk < gridDim.x; kk++){
    if (abs_row_idx*N + kk*TILEDIM_M + tx < N*N){
      As[ty*TILEDIM_M + tx] = A[abs_row_idx*N + kk*TILEDIM_M + tx];
    }
    else{
      As[ty*TILEDIM_M + tx] = 0;
    }
    if ((kk*TILEDIM_M+ty)*N + abs_col_idx < N*N){  
      Bs[ty*TILEDIM_M + tx] = B[(kk*TILEDIM_M+ty)*N + abs_col_idx];
    }
    else{
      Bs[ty*TILEDIM_M + tx] = 0;
    }
    __syncthreads();

    for (int k=0; k<TILEDIM_M; k++)
      Cij += As[ty*TILEDIM_M + k] * Bs[k * TILEDIM_M + tx];

    __syncthreads();
  }
  if((abs_row_idx < N) && (abs_col_idx < N)){
    C[abs_row_idx*N + abs_col_idx] = Cij;
  }
} */

__global__ void matMul(int N, _FTYPE_ *C, _FTYPE_ *A, _FTYPE_ *B) {
int ty = threadIdx.y;
  int tx = threadIdx.x;
  int by = blockIdx.y;
  int bx = blockIdx.x;

  int abs_row_idx =  by*blockDim.y + ty;
  int abs_col_idx =  bx*blockDim.x + tx;

  _FTYPE_ * __restrict__ As = &smem[0];
  _FTYPE_ * __restrict__ Bs = &smem[TILEDIM_M * TILEDIM_N];
  register _FTYPE_ Cij[TOTAL_WORK] = {0};

  if((abs_row_idx < N) && (abs_col_idx < N)){
  #pragma unroll
  for (int a = by*TILEDIM_M*N, b = bx*TILEDIM_N; a < N + by*TILEDIM_M*N; a += TILEDIM_N, b += TILEDIM_N*N) {
    int i;
    if (N % TILEDIM_N == 0){
      #pragma unroll
      for (i = 0; i < NUM_WORK_Y ; ++i) {
        #pragma unroll
        for(int j = 0; j < NUM_WORK_X; j++){
          As[(ty + i*BLOCK_DOM_Y)*TILEDIM_N + tx + j * BLOCK_DOM_X] = A[a + (ty + i*BLOCK_DOM_Y)*N + tx + j * BLOCK_DOM_X];
          Bs[(ty + i*BLOCK_DOM_Y)*TILEDIM_N + tx + j * BLOCK_DOM_X] = B[b + (ty + i*BLOCK_DOM_Y)*N + tx + j * BLOCK_DOM_X]; 
        }
      }
    }
    else{
      for (int i = 0; i < NUM_WORK_Y; ++i) {
        for(int j = 0; j < NUM_WORK_X; j++){
          int ity = ty + i * BLOCK_DOM_Y;

          int itx = tx + j * BLOCK_DOM_X;
          if ((a + ity * N >= N * N) || (a + itx >= (by * TILEDIM_M + 1) * N)) {
              As[ity * TILEDIM_N + itx] = 0;
          } else {
              As[ity * TILEDIM_N + itx] = A[a + ity * N + itx];
          }

          if ((b + ity * N >= N * N) || (bx * TILEDIM_N + itx >= N)) {
              Bs[ity * TILEDIM_N + itx] = 0;
          } else {
              Bs[ity * TILEDIM_N + itx] = B[b + ity * N + itx];
          }
        }
    }


    }

    __syncthreads(); 
    for (int k = 0; k < TILEDIM_N; k++) {
      #pragma unroll
      for (i = 0; i < NUM_WORK_Y; ++i) {
        #pragma unroll
        for(int j = 0; j < NUM_WORK_X; j++){
          Cij[i + j * NUM_WORK_Y] += As[(ty + i*BLOCK_DOM_Y)*TILEDIM_N + k] * Bs[k*TILEDIM_N + tx + j * BLOCK_DOM_X];
        }
      }
    }
    __syncthreads(); 
  }
    int c = N*TILEDIM_M*by + TILEDIM_N*bx;
    int row_idx = TILEDIM_M*by+ty;
    int col_idx = TILEDIM_N*bx+tx;
    
    if (N % TILEDIM_N != 0)
    {
      #pragma unroll
      for(int j = 0; j < NUM_WORK_X; j++){
        if (col_idx + j * BLOCK_DOM_X<N){
          #pragma unroll
          for (int i = 0; i < NUM_WORK_Y; ++i) {
            if (row_idx + i*BLOCK_DOM_Y < N){
              C[c + N * (ty + i*BLOCK_DOM_Y) + tx + j * BLOCK_DOM_X] = Cij[j*NUM_WORK_Y+i];
            }
          }
        }
      }
    }
    else{
      #pragma unroll
      for (int i = 0; i < NUM_WORK_Y; ++i) {
        for(int j = 0; j < NUM_WORK_X; j++){
          C[c + N * (ty + i*BLOCK_DOM_Y) + tx + j * BLOCK_DOM_X] = Cij[j * NUM_WORK_Y + i];
        }
      }
    }
  }
    
}

  



#endif
